#include "hip/hip_runtime.h"
/* CUDA kernel template for parameter sweeping 

    - striding requires attention!

    - hist contains only one state variable, cvar, while X contains all of them
        between the node and parsweep dims

    - the mapping between launch configuration and parameter space grid should be
        done more systematically 

*/

inline __device__ int wrap(int i) {

    if (i >= 0)
        return i % $horizon;
    else
        if (i == - $horizon)
            return 0;
        else
            return $horizon + (i % $horizon);

}

// begin defining models
#define defmodel(name, X, pars, n_thr, par_ij, input) inline __device__ void name\
    (float * __restrict__ X, void * __restrict__ pars, int n_thr, int par_ij, float input)

defmodel(bistable_euler, Y, p, nt, pi, i)
{
    float x   = Y[nt*0 + pi]
       ,  exc = *((float*) p)
       ,  dx  = (x - x*x*x/3.0)/20.0 + i + exc;

    Y[nt*0 + pi] = x + $dt*dx;
}

defmodel(fhn_euler, X, pars, nt, pi, in)
{
    float x = X[nt*0 + pi]
        , y = X[nt*1 + pi]
        , a = *((float*) pars)

        , dx = (x - x*x*x/3.0 + y)*3.0
        , dy = (a - x + in)/3.0;

    X[nt*0 + pi] = x + $dt*dx;
    X[nt*1 + pi] = y + $dt*dy;
}

#undef defmodel
// end model definitions

__global__ void kernel(int step, int * __restrict__ idel, 
                       float * __restrict__ hist, 
                       float * __restrict__ conn, 
                       float * __restrict__ X
                       )
{

    int par_i  = blockIdx.x
      , par_j  = threadIdx.x
      , par_ij = blockDim.x*par_i + par_j
      , n_thr  = blockDim.x*gridDim.x
      , hist_idx
      ;

    float input
      , gsc = $gsc0 + par_i*$dgsc
      , exc = $exc0 + par_j*$dexc
      ;

    for (int i=0; i<$n; i++)
    {
        input = 0.0;

        for (int j=0; j<$n; j++) {

                    //   stride*index
            hist_idx = $n*n_thr*wrap(step - 1 - idel[j*$n + i])  // step
                     +    n_thr*i                                // node index 
                     +        1*par_ij;                          // parsweep index

            input += conn[j*$n + i]*hist[hist_idx];
        }

        input *= gsc/$n;

        $model(X + n_thr*$nsv*i, &exc, n_thr, par_ij, input);
    }
}

// update history
__global__ void update(int step, float *hist, float *X)
{
    int par_ij = threadIdx.x
      , n_thr  = blockDim.x
      ;

    for (int i=0; i<$n; i++)
        hist[n_thr*$n*wrap(step) + n_thr*i + par_ij] = X[$nsv*n_thr*i + n_thr*$cvar*0 + par_ij];
}

