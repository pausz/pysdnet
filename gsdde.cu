#include "hip/hip_runtime.h"
/*

CUDA kernel template for integrating an SDDE.

template params: threadid, N, horizon, k, dt

functions

    - wrap() takes the step number and returns the corresponding step number
        assuming periodic boundaries.

    - step()

*/

inline __device__ int wrap(int i) {

    if (i >= 0)
        return i % $horizon;
    else
        if (i == - $horizon)
            return 0;
        else
            return $horizon + (i % $horizon);

}


__global__ void step(int * _i, // current step number/count
                     int * __restrict__ idelays, // delays in steps (N, N)
                     float * __restrict__ G, // coupling matrix (N, N)
                     float * __restrict__ hist, // history (horizon + 1, N)
                     float * __restrict__ randn) // randnums for this step (N,)

{

    int i = _i[0], j = $threadid;

    float xj, dxj, input;

    input = 0.0;
    for (int idx=0; idx<$N; idx++)
        input += G[j*$N + idx]*hist[$N*wrap(i - 1 - idelays[j*$N + idx]) + idx];

     xj = hist[$N*wrap(i - 1) + j];
    dxj = $dt*(  (xj - 5.0*pow((float)xj, 3.0f))/5.0 + $k*input/$N + randn[j]/5.0 );

    __threadfence();
    hist[$N*wrap(i) + j] = xj + dxj;

}


__global__ void get_state(int * _i, // current step no.
                          float * __restrict__ hist, // history
                          float * __restrict__ xout) // output

{
    int i = _i[0], j = $threadid;
    xout[j] = hist[$N*wrap(i) + j];
}

